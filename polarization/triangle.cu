#include "hip/hip_runtime.h"
/***************************************************************/
//
//Copyright (c) 2019 Esteban Egea-Lopez http://ait.upct.es/eegea
//
/**************************************************************/

//License from NVIDIA parts
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "../Common.h"
#include "../Complex.h"
#include "tracePolarizationFunctions.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
//#include <cmath>
using namespace optix;

//Launch variables
rtDeclareVariable(uint3, launchIndexTriangle, rtLaunchIndex, );
rtDeclareVariable(LPWavePayload, rayPayload, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(TriangleHit, ch_triangle_data, attribute triangle_hit_data, );

//Per-mesh local variables 
rtDeclareVariable(MaterialEMProperties, EMProperties, , );
rtDeclareVariable(uint, meshId, , );

//Penetration configuration
rtDeclareVariable(uint, usePenetration, , );
rtDeclareVariable(float, attenuationLimit, , );

RT_PROGRAM void closestHitTriangle()
{

	//Update payload
	const float rayLength = ch_triangle_data.geom_normal_t.w;
	const float3 hp= ray.origin + rayLength * ray.direction ;
	rayPayload.hitPoint =hp;
	const float3 gn=make_float3(ch_triangle_data.geom_normal_t.x,ch_triangle_data.geom_normal_t.y,ch_triangle_data.geom_normal_t.z);	
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,gn )); //Plane normal
	const float3 reflection_dir=reflect(ray.direction, n);
	const float aux=rayPayload.ndtd.w;
	rayPayload.ndtd = make_float4(reflection_dir); //initialized with float3, w is set to 0. and updated below
	//hash_combine_impl<uint>(rayPayload.refhash,ch_triangle_data.faceId);
	//rtPrintf("HASH \t%u\t%u\t%u\t%u\n",ch_triangle_date.faceId,rayPayload.reflections,rayPayload.hits,rayPayload.refhash);
	//Use reflections and hits to create hash
	hash_combine_impl<uint>(rayPayload.refhash,ch_triangle_data.faceId+rayPayload.reflections+rayPayload.hits);
	rayPayload.ndtd.w = aux+ rayLength;
	rayPayload.lrhpd =make_float4(hp); //lastReflectionHitPoint;
	rayPayload.lrhpd.w = rayPayload.ndtd.w; //totalDistanceTillLastReflection;
	
	
	
	//Compute reflection coefficient

	//Incidence angle (ai) is defined with respect to the surface, we use the complementary, which is 90-ai, and is the angle between ray and normal
	//WARNING: Assuming ray direction is normalized: dot(r,n)=cos(angle(r,n))
	//We use the fabs() because if a ray hits an internal face, the normal is reversed. The cos would be negative. For "closed" meshes this should not happen. However, in the borders, due to precision
	//it happens: a ray is not detected as hitting a face and gets inside the mesh, hitting an internal face later.
	//With penetration we can hit internal faces in closed meshes. This way, we also get the correct incidence angle again.
	
	//Compute local incidence coordinate system for reflection (components parallel and normal to the incidence plane)
	const float3 anorm_i=normalize(cross(ray.direction,n));
	const float3 apar_i=normalize(cross(anorm_i,ray.direction)); 
	//rtPrintf("\t%u\t%u\tn=(%.6e,%.6e,%.6e)|anorm_i|=(%.6e,%.6e,%.6e)=%.6e\t|apar_i|=(%.6e,%.6e,%.6e)=%.6e \n",launchIndexTriangle.x,launchIndexTriangle.y,n.x,n.y,n.z,anorm_i.x,anorm_i.y,anorm_i.z,length(anorm_i),apar_i.x,apar_i.y,apar_i.z,length(apar_i));


	
	//Reflected ray basis
	const float3 anorm_r=anorm_i; 
	const float3 apar_r=cross(anorm_r,reflection_dir); //Should change the direction with respect to the incidence parallel
	
	float cosA = fabs(dot(-ray.direction, n));


	//rtPrintf("G\t%u\t%u\t%d\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n", launchIndexTriangle.x, launchIndexTriangle.y, rayPayload.reflections, cosA, ray.direction.x, ray.direction.y, ray.direction.z, n.x, n.y, n.z, rayPayload.lrhpd.w);
	
	//Compute the reflection coefficients (only depend on incidence angle and dielectric properties)
	//Complex arithmetic: sum
	float2 argument = make_float2(EMProperties.dielectricConstant.x + (cosA*cosA) - 1.0f, EMProperties.dielectricConstant.y);
	float2 root = complex_sqrt(argument);
	
	
	//Soft reflection. 
	//Normal reflection coefficient (Electric field not in plane of incidence)
	const float2 Rnorm = complex_div(make_float2(cosA-root.x,-root.y),make_float2(cosA+root.x,root.y));


	//Reflection info log (to be used in external programs)
	//rtPrintf("H\t%u\t%u\t%d\t%d\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n", launchIndexTriangle.x, launchIndexTriangle.y, rayPayload.reflections, rayPayload.faceId, argument.x, argument.y, root.x, root.y, R.x, R.y, tR.x, tR.y);


	//Hard reflection.  
	//Parallel reflection coefficient (Electric field in plane of incidence)
	//float2 num = sca_complex_prod(cosA, make_float2(-EMProperties.dielectricConstant.x, -EMProperties.dielectricConstant.y));
	float2 num = sca_complex_prod(cosA, EMProperties.dielectricConstant);
	float2 div=num;

	//float2 div = sca_complex_prod(cosA, EMProperties.dielectricConstant);

//	num.x += root.x;
//	num.y += root.y;
//	div.x += root.x;
//	div.y += root.y;
	num -=root;
	div +=root;
	const float2	Rpar = complex_div(num, div);
	


		//Reflection info log (to be used in external programs)
	//	float mycos = dot(-ray.direction, n);
	//	rtPrintf("S\t%u\t%u\t%d\t%d\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n", launchIndexTriangle.x, launchIndexTriangle.y, rayPayload.reflections, rayPayload.faceId, argument.x, argument.y, root.x, root.y, R.x, R.y, tR.x, tR.y);
	//	rtPrintf("NN dot=%f angle=%f hp=(%f,%f,%f)per=(%f,%f)rayR=(%f,%f)\n",mycos,  acosf(mycos), rayPayload.hitPoint.x, rayPayload.hitPoint.y, rayPayload.hitPoint.z, EMProperties.dielectricConstant.x, EMProperties.dielectricConstant.y, rayPayload.prodReflectionCoefficient.x, rayPayload.prodReflectionCoefficient.y);

	

	//Get geometric  components, multiply by previous coefficients  and multiply by reflection coefficients computed above or transmission coefficients below
		
	//Geometric part normal
	const float2 Einorm=sca_complex_prod(dot(rayPayload.hor_v,anorm_i),rayPayload.hor_coeff) + sca_complex_prod(dot(rayPayload.ver_v,anorm_i),rayPayload.ver_coeff);
	//Geometric part parallel
	const float2 Eipar=sca_complex_prod(dot(rayPayload.hor_v,apar_i),rayPayload.hor_coeff) + sca_complex_prod(dot(rayPayload.ver_v,apar_i),rayPayload.ver_coeff);


	if ((usePenetration==1u) && (rayPayload.reflections<max_interactions)) {
		//Trace a penetration ray as a new ray. Recursive tracing, check stack depth>max_interactions
		//Quickly check for attenuation in dB, if att has a very low value we do not trace. Also, we do not want to overflow the float in the operations and get a nan.
		//Apply material attenuation. Again, we assume the material is not bending the ray in any direction
		
		
		//Typical values are -15 dB for 203 mm at 5 GHz => -75 dB/m
		//Considering real distance travelled through material
		float dbAtt=(EMProperties.tattenuation.x/cosA)*(EMProperties.tattenuation.y); //Attenuation in dB (power) distance*att/m = thickness/cosA*att/m
		
		//Considering that material has been travelled in perpendicular
		//float dbAtt=(EMProperties.tattenuation.x)*(EMProperties.tattenuation.y);
		float tAtt=rayPayload.accumulatedAttenuation + dbAtt; //Accumulate in log scale to avoid overflows
		if (tAtt>attenuationLimit) {
			//Copy payload
			LPWavePayload penPayload=rayPayload;
			penPayload.ndtd  = optix::make_float4(0, 0, 0, rayPayload.ndtd.w);
			penPayload.hits=rayPayload.hits+1;
			penPayload.flags = FLAG_NONE;
			//Assuming the media on both sides of the plane are the same (air, most likely), then the incidence angle is equal to the transmission angle, so the ray does not change trajectory
			//Otherwise, we have to rotate the ray by the transmission angle, where a_t (angle_transmission) and theta= 90-a_t, with respect to the vector ortoghonal to the normal and ray,
			//that is the normal vector of the plane defined by ray and mesh face normal.
			//We can use Rodrigues formula to rotate the vector given angle and unit vector e,  to avoid using rotating matrix
			//So, e = cross(normal,-ray), remember right hand rule and check this... already should be a unit vector since ray and normal are normalized
			//ray_rot=cos(theta)*ray + sin(theta)*(cross(e,ray)+(1-cos(theta)(dot(e,ray))e. 
			//Have to compute the cos and sin of theta=90-a_
			//Check the above
			//Assume equal media
			//Transmission coefficient (1+reflection coefficient)
			//
			//New horizontal (normal)  coefficient
			penPayload.hor_coeff=complex_prod(Einorm,make_float2(1.0f+Rnorm.x,Rnorm.y));
			//New vertical (parallel)  coefficient
			penPayload.ver_coeff=complex_prod(Eipar,make_float2(1.0f+Rpar.x,Rpar.y));

			//Update vectors. Assuming they are equal to incident vectors
			penPayload.ver_v=apar_i;
			penPayload.hor_v=anorm_i;


			//penPayload.prodReflectionCoefficient = complex_prod(rayPayload.prodReflectionCoefficient,make_float2(1.0f+R.x, R.y)); 
			//Attenuation
			penPayload.accumulatedAttenuation = tAtt;
			//rtPrintf("AT cosA=%f\tatt=%f\ttAtt=%f\nr=(%f,%f,%f)\tn=(%f,%f,%f)\n",cosA,dbAtt,tAtt,ray.direction.x,ray.direction.y,ray.direction.z,n.x,n.y,n.z);
			traceLPReflection(penPayload, rayPayload.hitPoint, ray.direction,launchIndexTriangle.x,launchIndexTriangle.y);
		}
	}
	//Update here reflection coefficient, otherwise we multiply reflection and transmission in the transmission above

	//New horizontal (normal)  coefficient
	rayPayload.hor_coeff=complex_prod(Einorm,Rnorm);
	//New vertical (parallel)  coefficient
	rayPayload.ver_coeff=complex_prod(Eipar,Rpar);
	
	
	
	//if (launchIndexTriangle.x==1660 && launchIndexTriangle.y==0) { //Use setPrintLaunchIndex for this instead of if
	rtPrintf("T\t%u\t%u\tRnorm(%.6e,%.6e)=Einorm=(%.6e,%.6e)Rpar=(%.6e,%.6e)Eipar=(%.6e,%.6e)\n",launchIndexTriangle.x,launchIndexTriangle.y,Rnorm.x,Rnorm.y,Einorm.x,Einorm.y,Rpar.x,Rpar.y,Eipar.x,Eipar.y);
	rtPrintf("T\t%u\t%u\thc=(%.6e,%.6e)hn=%.6evn=%.6evc=(%.6e,%.6e)vp=%.6e ver_v=(%.6e,%.6e,%.6e)\n",launchIndexTriangle.x,launchIndexTriangle.y,rayPayload.hor_coeff.x,rayPayload.hor_coeff.y,dot(rayPayload.hor_v,anorm_i),dot(rayPayload.ver_v,anorm_i),rayPayload.ver_coeff.x,rayPayload.ver_coeff.y,dot(rayPayload.ver_v,apar_i), rayPayload.ver_v.x,rayPayload.ver_v.y,rayPayload.ver_v.z);
	rtPrintf("T\t%u\t%u\t|anorm_r|=(%.6e,%.6e,%.6e)=%.6e\t|apar_r|=(%.6e,%.6e,%.6e)=%.6e \n",launchIndexTriangle.x,launchIndexTriangle.y,anorm_r.x,anorm_r.y,anorm_r.z,length(anorm_r),apar_r.x,apar_r.y,apar_r.z,length(apar_r));
	//}
	
	//Update vectors
	rayPayload.ver_v=apar_r;
	rayPayload.hor_v=anorm_r;
	++rayPayload.reflections;

}



//For Optix 5.x
//Mesh buffers
rtBuffer<float3> vertex_buffer;
rtBuffer<int3>   index_buffer;
rtBuffer<uint> faceId_buffer;

rtDeclareVariable(TriangleHit, int_triangle_data, attribute triangle_hit_data, );

RT_PROGRAM void intersectTriangle(int primIdx)
{
	const int3 v_idx = index_buffer[primIdx];

	const float3 p0 = vertex_buffer[v_idx.x];
	const float3 p1 = vertex_buffer[v_idx.y];
	const float3 p2 = vertex_buffer[v_idx.z];

	// Intersect ray with triangle
	float3 normal;
	float  t, beta, gamma;

	//rtPrintf("PreIntersection idx=%d ray=(%f,%f,%f)", primIdx, ray.direction.x, ray.direction.y, ray.direction.z);
	if (intersect_triangle(ray, p0, p1, p2, normal, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			TriangleHit h;
			h.triId = primIdx;
			h.geom_normal_t = make_float4(normal.x,normal.y,normal.z,t);
			h.faceId = faceId_buffer[primIdx];


			int_triangle_data = h;
			//rtPrintf("Intersection idx=%d ray=(%f,%f,%f)", primIdx, ray.direction.x, ray.direction.y, ray.direction.z);
			rtReportIntersection( /*material index*/ 0);
		}
	}
}
RT_PROGRAM void boundsTriangle(int primIdx, float result[6])
{
	const int3 v_idx = index_buffer[primIdx];

	const float3 p0 = vertex_buffer[v_idx.x];
	const float3 p1 = vertex_buffer[v_idx.y];
	const float3 p2 = vertex_buffer[v_idx.z];

	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->m_min = fminf(fminf(p0, p1), p2);
	aabb->m_max = fmaxf(fmaxf(p0, p1), p2);
}
