#include "hip/hip_runtime.h"
/***************************************************************/
//
//Copyright (c) 2019 Esteban Egea-Lopez http://ait.upct.es/eegea
//
/**************************************************************/


#include "../../Common.h"
#include "../../traceFunctions.h"
#include "linearPolarizationFunctions.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
using namespace optix;


//Generation of ray sphere


//Ray Sphere buffer
rtBuffer<float3, 2> raySphere2D;

//Transmitter buffer
rtBuffer<Transmitter, 1> txBuffer;

//Launch variables
rtDeclareVariable(uint3, launchIndex, rtLaunchIndex, );



//Configuration variables
rtDeclareVariable(uint2, raySphereSize, , );
rtDeclareVariable(uint, usePenetration, , );

RT_PROGRAM void genRayAndReflectionsFromSphereIndex()
{


	//3D kernel launch [elevation, azimuth, transmitters]	

	uint2 idx = make_uint2(launchIndex.x, launchIndex.y); //[elevation, azimuth]
	//index goes from 0 to raySphereSize.x-1 //The last elevation step corresponds to 180 degrees elevation
	if ((idx.x == 0 ||idx.x==  raySphereSize.x-1  ) && idx.y != 0) {
		//These rays are all the same (0,1,0) or (0,-1,0). Only trace  (0,0) and (last,0) corresponding to 0 and 180 elevation degrees
		return;
	}

	const Transmitter tx = txBuffer[launchIndex.z];
	
	float3 origin = tx.origin;
	
	float3 ray_direction = raySphere2D[idx];

	LPWavePayload rayPayload;
	rayPayload.ndtd = optix::make_float4(0.0f);
	//rayPayload.hitPoint = origin;
	rayPayload.hitPointAtt =make_float4(origin);
	rayPayload.hitPointAtt.w=0.0f;

	rayPayload.hor_coeff=make_float2(1.0f,0.0f);	
	rayPayload.ver_coeff=make_float2(1.0f,0.0f);	

	fillPolarization(rayPayload,tx.polarization, ray_direction);
	
	//rtPrintf("\t%u\t%u\tray=(%f,%f,%f),pol=(%f,%f,%f), polt=(%f,%f,%f)\n",launchIndex.x, launchIndex.y,ray_direction.x,ray_direction.y,ray_direction.z,tx.polarization.x,tx.polarization.y,tx.polarization.z,rayPayload.E.x,rayPayload.E.y,rayPayload.E.z);
	//rtPrintf("G\t%u\t%u\tray=(%f,%f,%f),pol=(%f,%f,%f),\n",launchIndex.x, launchIndex.y,ray_direction.x,ray_direction.y,ray_direction.z,tx.polarization.x,tx.polarization.y,tx.polarization.z);
	//rtPrintf("G\t%u\t%u\thor_v=(%.6e,%.6e,%.6e),ver_v=(%.6e,%.6e,%.6e), hor_coeff=(%.6e,%.6e), ver_coeff(%.6e,%.6e)\n",launchIndex.x, launchIndex.y,rayPayload.hor_v.x,rayPayload.hor_v.y,rayPayload.hor_v.z,rayPayload.ver_v.x,rayPayload.ver_v.y,rayPayload.ver_v.z, rayPayload.hor_coeff.x,rayPayload.hor_coeff.y,rayPayload.ver_coeff.x,rayPayload.ver_coeff.y);
	
	rayPayload.lrhpd = make_float4(origin);
	rayPayload.lrhpd.w = 0.0f; //totalDistanceTillLastReflection
	rayPayload.electricFieldAmplitude = 1.0f; //Normalized Eo=1. Antenna Gain = 1. TODO: Implement antenna gain with buffer dependent on the ray direction and txId : initialEFAmplitude[txId] * antennaGain[txId]);
	//rayPayload.accumulatedAttenuation=0.0f;
	rayPayload.rhfr=make_uint4(0u,0u,FLAG_NONE,0u);
//	rayPayload.reflections = 0;
//	rayPayload.hits = 0;
//	rayPayload.flags= FLAG_NONE;
//
//	rayPayload.refhash=0;
#ifdef OPAL_LOG_TRACE
	rayPayload.initialRayDir=ray_direction;
#endif
	//Print all rays generated
	//rtPrintf("A\t%u\t%u\t%f\t%f\t%f\n", launchIndex.x, launchIndex.y, ray_direction.x, ray_direction.y, ray_direction.z);

	traceReflection<LPWavePayload>(rayPayload, OPAL_RAY_REFLECTION, origin, ray_direction, launchIndex.x,launchIndex.y);
	//traceLPReflection(rayPayload, origin, ray_direction, launchIndex.x,launchIndex.y);


}


