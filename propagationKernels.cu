#include "hip/hip_runtime.h"


#include "Common.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
using namespace optix;



//Ray Sphere buffer
rtBuffer<float3> raySphere;
rtBuffer<float3, 2> raySphere2D;
rtBuffer<TriangleHit, 2>  hits;

//rtBuffer<int, 1> init;
rtBuffer<Transmitter,1> tx_origin;


//Receiver buffers
//rtBuffer<rtBufferId<uint, 3>, 1> duplicatesBuffer; //Filter duplicates buffer [[elevationBlockSize, azimuthBlockSize, transmitter],receiver ]


rtBuffer<rtBufferId<int, 3>, 1> internalRaysBuffer; //Filter internal rays buffer  [[elevationSteps, azimuthSteps, transmitter],receiver]

rtBuffer<uint, 2> facesBuffer; //Face-based duplicates [faceId,receiver]
rtBuffer<int, 3> bufferMinD; //minDistance ray to receiver [reflections, faceId,receiver];
rtBuffer<DuplicateReflection, 3> bufferMinE; //Buffer with the Electric field of the minimum distance ray [reflections, faceId,receiver];
rtBuffer<ReceptionInfo, 2> receptionInfoBuffer; //Results buffer [receiver,transmitter]



//rtDeclareVariable(uint2, duplicateBlockSize, , );

rtDeclareVariable(uint3, launchIndex, rtLaunchIndex, );

rtDeclareVariable(rtObject, staticMeshes, , );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(float, min_t_epsilon, , );
rtDeclareVariable(unsigned int, max_interactions, , );
rtDeclareVariable(uint2, tx_rx, , ); //[transmitters, receivers]
rtDeclareVariable(unsigned int, number_of_faces, , );

rtDeclareVariable(uint2, raySphereSize, , );
//rtDeclareVariable(uint, initialize, , );

/*RT_PROGRAM void initializeBuffers() {
	
	if (launchIndex == make_uint3(0, 0, 0)) {
		for (unsigned int i = 0; i < tx_rx.y; ++i)
		{
			for (unsigned int j = 0; j < tx_rx.x; ++j)
			{
				uint2 index = make_uint2(i, j);

				//rtPrintf("Initializing reception buff  Ep0[%u,%u]=(%f,%f)  \n", index.x, index.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y);
				receptionInfoBuffer[index].sumRxElectricField = make_float2(0.0f, 0.0f);
				receptionInfoBuffer[index].directHits = 0;
				receptionInfoBuffer[index].reflections = 0;
				//rtPrintf("Initializing reception buff  Ep0[%u,%u]=(%f,%f)  \n", index.x, index.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y);



			}

		}
	}
	//Each launch initializes its element in the receivers buffer
	//rtPrintf("Initializing buffers tx_rx=(%u,%u)\n", tx_rx.x, tx_rx.y);


	for (unsigned int i = 0; i < tx_rx.y; ++i)
	{	
		if (launchIndex.y < 180) {
			rtBufferId<uint, 3>& db = duplicatesBuffer[i];
			uint3 iddb = make_uint3(launchIndex.x / duplicateBlockSize.x, launchIndex.y / duplicateBlockSize.y, launchIndex.z);
			db[iddb] = 0u;
		}
		rtBufferId<int, 3>& ib = internalRaysBuffer[i];
		ib[launchIndex] = -1;
		//rtPrintf("i=%u db[%u,%u,%u]=%u ib=%d \n", i, launchIndex.x, launchIndex.y, launchIndex.z, db[launchIndex], ib[launchIndex]);

	}

}
*/
RT_PROGRAM void initializeBuffersFaceBased() {
	//rtPrintf("Initializing reception buff  tx_rx=(%u,%u)\n", tx_rx.x, tx_rx.y);
	if (launchIndex == make_uint3(0, 0, 0)) {
		
		for (unsigned int i = 0; i < tx_rx.y; ++i)
		{
			
			for (unsigned int j = 0; j < tx_rx.x; ++j)
			{
				uint2 index = make_uint2(i, j);
				//rtPrintf("Initializing reception buff  index=(%u,%u)\n", index.x, index.y);
				//rtPrintf("Initializing reception buff  Ep0[%u,%u]=(%f,%f)  \n", index.x, index.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y);
				receptionInfoBuffer[index].sumRxElectricField = make_float2(0.0f, 0.0f);
				receptionInfoBuffer[index].directHits = 0;
				receptionInfoBuffer[index].reflections = 0;
				//rtPrintf("Initializing reception buff  Ep0[%u,%u]=(%f,%f)  \n", index.x, index.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y);



			}
			for (unsigned int k = 0; k < number_of_faces; ++k)
			{
				uint2 idf = make_uint2(k, i);
				facesBuffer[idf] = 0u;
				for (unsigned int  l = 0; l < max_interactions; ++l) 
				{
					
					uint3 idmd = make_uint3(l, k, i);
					bufferMinD[idmd] = 2147483647;
					bufferMinE[idmd].E = make_float2(0.0f, 0.0f);
					bufferMinE[idmd].r = 0;
				//	rtPrintf("Initializing faces buff idm=(%u,%u,%u)=%d \n", idmd.x,idmd.y,idmd.z, bufferMinD[idmd]);

					
				}
			}

		}
	}
	//Each launch initializes its element in the receivers buffer
	//rtPrintf("Initializing buffers tx_rx=(%u,%u)\n", tx_rx.x, tx_rx.y);


	for (unsigned int i = 0; i < tx_rx.y; ++i)
	{
		
		rtBufferId<int, 3>& ib = internalRaysBuffer[i];
		ib[launchIndex] = -1;
		//rtPrintf("i=%u db[%u,%u,%u]=%u ib=%d \n", i, launchIndex.x, launchIndex.y, launchIndex.z, db[launchIndex], ib[launchIndex]);

	}

}
RT_PROGRAM void genRayAndReflectionsFromSphereIndex()
{

	

		uint2 idx = make_uint2(launchIndex.x, launchIndex.y); //[elevation, azimuth]
		//index goes from 0 to raySphereSize.x-1 //The last elevation step corresponds to 180 degrees elevation
		if ((idx.x == 0 ||idx.x==  raySphereSize.x-1  ) && idx.y != 0) {
			//These rays are all the same (0,1,0) or (0,-1,0). Only trace  (0,0) and (last,0) corresponding to 0 and 180 elevation degrees
			return;
		}
		float3 origin = tx_origin[launchIndex.z].origin;
		float3 ray_direction = raySphere2D[idx];

		EMWavePayload rayPayload;
		rayPayload.geomNormal = optix::make_float3(0, 0, 0);
		rayPayload.nextDirection = optix::make_float3(0, 0, 0);
		rayPayload.hitPoint = origin;
		rayPayload.polarization = tx_origin[launchIndex.z].polarization;
		rayPayload.electricFieldAmplitude = 1.0f; //Normalized Eo=1. Antenna Gain = 1. Implement antenna gain with antennaBuffer dependent on the ray direction and txId(launchIndex) : initialEFAmplitude[launchIndex.z] * antennaGain[launchIndex]);
		rayPayload.t = -1.0f;
		rayPayload.reflections = 0;

		rayPayload.hits = 0;
		rayPayload.totalDistance = 0.0f;
		rayPayload.end = false;

		rayPayload.prodReflectionCoefficient = make_float2(1.0f, 0.0f);
		rayPayload.faceId = 0u;



		//rtPrintf("Lsphere2D el=%u az=%u tx=%u ray=(%f,%f,%f)\n", launchIndex.x, launchIndex.y, launchIndex.z,ray_direction.x, ray_direction.y, ray_direction.z );


		// Each iteration is a segment (due to reflections) of the ray path.  The closest hit will
		// return new segments to be traced here. Additionally, the closest hit at receiver will generate another ray to continue the propagation
		int i = 0;
		//int lastReflectionNumber = 0;
		//rtPrintf("Generating ray i.x=%u i.y=%u, ray=(%f,%f,%f) inter=%d end=%d \n", launchIndex.x, launchIndex.y, ray_direction.x, ray_direction.y, ray_direction.z, rayPayload.reflections, rayPayload.end);
		//rtPrintf("A\t%u\t%u\t%f\t%f\t%f\n", launchIndex.x, launchIndex.y, ray_direction.x, ray_direction.y, ray_direction.z);
		
		while (true) {
			optix::Ray myRay(origin, ray_direction, 0, min_t_epsilon, RT_DEFAULT_MAX);

			rtTrace(root, myRay, rayPayload);
			i++;
			//Miss or too much attenuation
			if (rayPayload.end) {
				break;
			}
			//Max number of reflections
			if (rayPayload.reflections > max_interactions) {
				break;
			}
			//Hit one or more receivers and no meshes
			/*if (rayPayload.reflections == 0 && rayPayload.hits > 0) {
				//Continue the ray
				origin = rayPayload.hitPoint;
				ray_direction = rayPayload.nextDirection;
				rtPrintf("1 Continuing ray i.x=%u i.y=%u, reflections=%d hits=%d rd=(%f,%f,%f) origin=(%f,%f,%f) end=%d \n", launchIndex.x, launchIndex.y, rayPayload.reflections, rayPayload.hits, ray_direction.x, ray_direction.y, ray_direction.z, rayPayload.hitPoint.x, rayPayload.hitPoint.y, rayPayload.hitPoint.z, rayPayload.end);
				continue;
			}
			//No additional reflection in the last trace, it may have hit a receiver, otherwise it would have been killed by miss program. Now kill
			if (rayPayload.reflections == lastReflectionNumber  && rayPayload.hits > 0) {
				//Continue the ray
				origin = rayPayload.hitPoint;
				ray_direction = rayPayload.nextDirection;
				rtPrintf("2 Continuing ray i.x=%u i.y=%u, reflections=%d hits=%d rd=(%f,%f,%f) origin=(%f,%f,%f) end=%d \n", launchIndex.x, launchIndex.y, rayPayload.reflections, rayPayload.hits, ray_direction.x, ray_direction.y, ray_direction.z, rayPayload.hitPoint.x, rayPayload.hitPoint.y, rayPayload.hitPoint.z, rayPayload.end);

				continue;

			}*/
			//Reflection or going through receiver
			//lastReflectionNumber = rayPayload.reflections;
			// Update ray data for the next path segment
			ray_direction = rayPayload.nextDirection;
			origin = rayPayload.hitPoint;
			//rtPrintf("Continuing or reflecting ray i.x=%u i.y=%u, reflections=%d hits=%d rd=(%f,%f,%f) origin=(%f,%f,%f) end=%d \n", launchIndex.x, launchIndex.y, rayPayload.reflections, rayPayload.hits, ray_direction.x, ray_direction.y, ray_direction.z, rayPayload.hitPoint.x, rayPayload.hitPoint.y, rayPayload.hitPoint.z, rayPayload.end);
			
			//Reflection info log (to be used in external programs)
			//rtPrintf("R\t%u\t%u\t%u\t%d\t%d\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n", launchIndex.x, launchIndex.y, launchIndex.z, rayPayload.reflections, rayPayload.hits, ray_direction.x, ray_direction.y, ray_direction.z, rayPayload.hitPoint.x, rayPayload.hitPoint.y, rayPayload.hitPoint.z, rayPayload.totalDistance);

			//if (rayPayload.reflections == 0) {
				//rtPrintf("Generating reflection with 0 reflection i.x=%u i.y=%u, inter=%d end=%d \n", launchIndex.x, launchIndex.y, rayPayload.reflections, rayPayload.end);
				//break;
			//}
			//rtPrintf("Reflecting ray i.x=%u i.y=%u, inter=%d hits=%d rd=(%f,%f,%f) origin=(%f,%f,%f) end=%d \n", launchIndex.x, launchIndex.y, rayPayload.reflections, rayPayload.hits, rayPayload.reflectionDirection.x, rayPayload.reflectionDirection.y, rayPayload.reflectionDirection.z, rayPayload.hitPoint.x, rayPayload.hitPoint.y, rayPayload.hitPoint.z, rayPayload.end);
		}

	
}









//rtDeclareVariable(TriangleHit, hit_attr, attribute hit_attr, );



rtDeclareVariable(SphereHit, hit_attr, attribute hit_attr, );
rtDeclareVariable(EMWavePayload, hitPayload, rtPayload, );
rtDeclareVariable(float, k, , );
rtDeclareVariable(uint3, receiverLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(uint, receiverId, , ); //Buffer id
rtDeclareVariable(uint, externalId, , ); //External id  used to identify receivers 


rtDeclareVariable(float4, sphere, , );
rtDeclareVariable(optix::Ray, ray_receiver, rtCurrentRay, );

//rtDeclareVariable(int, duplicateStep, , );



rtDeclareVariable(rtCallableProgramId<float2(float2)>, complex_exp_only_imaginary, , );
rtDeclareVariable(rtCallableProgramId<float2(float, float2)>, sca_complex_prod, , );
rtDeclareVariable(rtCallableProgramId<float2(float2, float2)>, complex_prod, , );


/*RT_PROGRAM void closestHitReceiver()
{

	//TODO: We do not check polarization between tx and rx. Can be done comparing payload polarization and receiver polarization

	//Do not end the ray, it can pass the reception sphere and reflect on a wall, inside or outside the receiver sphere
	//Store for use later
	float prevTd = hitPayload.totalDistance;

	//Update ray data
	hitPayload.totalDistance += hit_attr.t;
	hitPayload.hitPoint = ray_receiver.origin + hit_attr.t*ray_receiver.direction;
	hitPayload.nextDirection = ray_receiver.direction;

	//Check if ray is hitting his own tx (transmitter are also receiver usually) A transmitter cannot receive while it is transmitting, unless other channel is used.
	uint2 index = make_uint2(receiverId, receiverLaunchIndex.z); 
	if (externalId == receiverLaunchIndex.z) {
		//Outgoing ray
		//rtPrintf("External. id=%u tx=%u i.x=%u i.y=%u, ray=(%f,%f,%f) origin=(%f,%f,%f) t=%f rId=%d\n", index.x, index.y, receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, ray_receiver.origin.x, ray_receiver.origin.y, ray_receiver.origin.z, hit_attr.t, receiverId);

		return;
	}


	//uint2 index = make_uint2(receiverId, receiverLaunchIndex.z);//Index is given by transmitter and receiver ids
	int reflections = hitPayload.reflections;
	
	//rtPrintf("Hit. id=%u tx=%u i.x=%u i.y=%u, ray=(%f,%f,%f) origin=(%f,%f,%f) t=%f rId=%d\n", index.x, index.y, receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, ray_receiver.origin.x, ray_receiver.origin.y, ray_receiver.origin.z, hit_attr.t, receiverId);
	//rtPrintf("   hitpoint=(%f,%f,%f) rId=%d\n", hitPayload.hitPoint.x, hitPayload.hitPoint.y, hitPayload.hitPoint.z, receiverId);

	//rtPrintf("DR. Direct hit  Ep=(%f,%f)  rId=%d \n", receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);




	//Check incoming or outgoing ray
	rtBufferId<int, 3>& ib = internalRaysBuffer[receiverId];
	int prevRef = ib[receiverLaunchIndex];
	if (prevRef < 0) {
		//Incoming ray. Store number of reflections
		ib[receiverLaunchIndex] = reflections;
		++hitPayload.hits;
	}
	else {
		//Outgoing ray. Check reflections
		//Reinit the buffer
		ib[receiverLaunchIndex] = -1;
		if (prevRef == reflections) {
			//Ray has not been reflected within the receiver sphere, ignore it
			//rtPrintf("  rId=%d----> outgoing ray \n", receiverId);

			return;

		}
	}
	//float r = hit_attr.t; //Distance to origin
	//rtPrintf("HR. rx=(%f,%f,%f) radius=%f id=%u tx=%u\n", sphere.x,sphere.y,sphere.z, sphere.w, index.x, index.y);
	//rtPrintf("HR. HitPayload inte=%d \n", hitPayload.reflections);
	

	
	if (reflections == 0) {
		//This is a direct ray

		int old = atomicAdd(&receptionInfoBuffer[index].directHits, 1);
		//rtPrintf("before dh=%d,id=%u tx=%u receptionInfoBuffer[index].directHits=%d  i.x=%u i.y=%u, ray=(%f,%f,%f) t=%f \n", old, index.x, index.y, receptionInfoBuffer[index].directHits, receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, hit_attr.t);
		if (old > 0) {
			//rtPrintf("HR. already direct \n");
			//Already has a direct ray counted, just ignore
			return;
		}
		//Compute electric field. For direct rays, the distance is always between tx and rx
		float3 prx = make_float3(sphere.x, sphere.y, sphere.z);
		float3 ptx = tx_origin[launchIndex.z].origin;
		float d = length(prx - ptx);
		//rtPrintf("DR. Direct hit   i.x=%u i.y=%u ray=(%f,%f,%f) prx=(%f,%f,%f) ptx=(%f,%f,%f) rId=%d \n",  receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, prx.x, prx.y, prx.z, ptx.x, ptx.y, ptx.z, receiverId);
		//rtPrintf("HR. Ep.x=%f Ep.y=%f Eo=%f\n", receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, hitPayload.electricFieldAmplitude);
		float2 z = make_float2(0.0f, -k*d);
		//rtPrintf("HR. prx=(%f,%f,%f) k=%f d=%f prevTd=%f hitPayload.totalDistance=%f Eo=%f\n", prx.x, prx.y, prx.z, k, d, prevTd, hitPayload.totalDistance, hitPayload.electricFieldAmplitude);

		float2 zexp = complex_exp_only_imaginary(z);
		float2 E = sca_complex_prod((hitPayload.electricFieldAmplitude / d), zexp);
		
		float oldEx=atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.x, E.x);
		float oldEy=atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.y, E.y);
		//rtPrintf("DR. Direct hit   i.x=%u i.y=%u  Ep=(%f,%f) E=(%f,%f) En=(%f,%f) rId=%d \n", receiverLaunchIndex.x, receiverLaunchIndex.y, oldEx,oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);
		//Direct hit info log (to be used in external programs)
		rtPrintf("DH\t%u\t%u\t%u\t%f\t%f\t%f\t%f\t%f\t%f\t%d\n", receiverLaunchIndex.x, receiverLaunchIndex.y, receiverLaunchIndex.z, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);

		//receptionInfoBuffer[index].hasDirectContribution = true;
		//rtPrintf("%f\t%f\n", E.x, E.y);
	}
	else {

			//Reflected ray
			//Check for duplicates
			//Index of buffer is receiverLaunchIndex= receiverLaunchIndex.x, receiverLaunchIndex.y,txId
		rtBufferId<uint, 3>& db = duplicatesBuffer[receiverId];
		uint azimuth = receiverLaunchIndex.y;
		if (azimuth >= 180) {
			azimuth -= 180;
		}
		uint3 idx = make_uint3(receiverLaunchIndex.x / duplicateBlockSize.x, azimuth / duplicateBlockSize.y, receiverLaunchIndex.z);

		//Represent the number of reflections with a bit position in a 32 bit unsigned integer. Up to 31 reflections. A ray with higher number of reflections should have been killed by the closest hit at mesh
		uint refBit = 0u;
		//Set bit at position (reflections) to 1, so for instance reflections=2 means ... ... ... 00000100 (32 bit unsigned)
		refBit |= 1u << hitPayload.reflections; 

		uint oldR = atomicOr(&db[idx], refBit); 
		//Check the value of the bit at position (reflections) 

		uint bit = (oldR >> hitPayload.reflections) & 1u; 

		//Bit already set for the block, consider it a duplicate
		if (bit == 1) {
		//	rtPrintf(" duplicated ray reflections=%d i.x=%u i.y=%u with idb.x=%u idb.y=%u oldR=%u bit=%u rId=%d\n", hitPayload.reflections, receiverLaunchIndex.x, receiverLaunchIndex.y, idx.x, idx.y,oldR, bit, receiverId);
			return;
		}

		//rtPrintf("Reflected hit reflections=%d i.x=%u i.y=%u db=%d with idb.x=%u idb.y=%u oldR=%u bit=%u rId=%d \n", hitPayload.reflections, receiverLaunchIndex.x, receiverLaunchIndex.y, db[receiverLaunchIndex], idx.x, idx.y, oldR, bit, receiverId);
		atomicAdd(&receptionInfoBuffer[index].reflections, 1);

			float3 prx = make_float3(sphere.x, sphere.y, sphere.z);
			float3 ptx = ray_receiver.origin;
			float d = length(prx - ptx);
			//Compute electric field
			//rtPrintf("ref totalDistance=%f d=%f reflections=%d i.x=%u i.y=%u \n", hitPayload.totalDistance, d, hitPayload.reflections, receiverLaunchIndex.x, receiverLaunchIndex.y);
			d += prevTd; //totalDistance

			float2 z = make_float2(0.0f, -k*d);
			float2 zexp = complex_exp_only_imaginary(z);
			float2 Rzexp = complex_prod(hitPayload.prodReflectionCoefficient, zexp);


			float2 E = sca_complex_prod((hitPayload.electricFieldAmplitude / d), Rzexp);
			//rtPrintf("ref R=(%f,%f) z=(%f,%f) zepx(%f,%f) Rzexp=(%f,%f), E=(%f,%f) i.x=%u i.y=%u \n", hitPayload.prodReflectionCoefficient.x, hitPayload.prodReflectionCoefficient.y, z.x, z.y, zexp.x, zexp.y, Rzexp.x, Rzexp.y, E.x, E.y, receiverLaunchIndex.x, receiverLaunchIndex.y);

			float oldEx = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.x, E.x);
			float oldEy = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.y, E.y);
			//rtPrintf("HR. i.x=%u i.y=%u  Reflected hit  reflections=%d Ep=(%f,%f) E=(%f,%f) En=(%f,%f) rId=%d \n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);

			//rtPrintf("Old E=(%f.%f) New E=(%f,%f) i.x=%u i.y=%u \n", oldx, oldy, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverLaunchIndex.x, receiverLaunchIndex.y);
			//rtPrintf("%f\t%f\n", E.x, E.y);
			//Reflected hit info log (to be used in external programs)
			rtPrintf("F\t%u\t%u\t%u\t%f\t%f\t%f\t%f\t%f\t%f\t%d\t%f\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y,  reflections, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId, prevTd, length(prx - ptx));

		
	}

}
*/

/*RT_PROGRAM void closestHitReceiverFaceBased()
{

	//TODO: We do not check polarization between tx and rx. Can be done comparing payload polarization and receiver polarization

	//Do not end the ray, it can pass the reception sphere and reflect on a wall, inside or outside the receiver sphere
	//Store for use later
	float prevTd = hitPayload.totalDistance;
	
	//Update ray data
	hitPayload.totalDistance += hit_attr.t;
	hitPayload.hitPoint = ray_receiver.origin + hit_attr.t*ray_receiver.direction;
	hitPayload.nextDirection = ray_receiver.direction;

	//Check if ray is hitting his own tx (transmitter are also receiver usually) A transmitter cannot receive while it is transmitting, unless other channel is used.
	uint2 index = make_uint2(receiverId, receiverLaunchIndex.z);
	if (externalId == receiverLaunchIndex.z) {
		//Outgoing ray
		//rtPrintf("External. id=%u tx=%u i.x=%u i.y=%u, ray=(%f,%f,%f) origin=(%f,%f,%f) t=%f rId=%d\n", index.x, index.y, receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, ray_receiver.origin.x, ray_receiver.origin.y, ray_receiver.origin.z, hit_attr.t, receiverId);

		return;
	}


	//uint2 index = make_uint2(receiverId, receiverLaunchIndex.z);//Index is given by transmitter and receiver ids
	int reflections = hitPayload.reflections;

	//rtPrintf("Hit. id=%u tx=%u i.x=%u i.y=%u, ray=(%f,%f,%f) origin=(%f,%f,%f) t=%f rId=%d\n", index.x, index.y, receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, ray_receiver.origin.x, ray_receiver.origin.y, ray_receiver.origin.z, hit_attr.t, receiverId);
	//rtPrintf("   hitpoint=(%f,%f,%f) rId=%d\n", hitPayload.hitPoint.x, hitPayload.hitPoint.y, hitPayload.hitPoint.z, receiverId);

	//rtPrintf("DR. Direct hit  Ep=(%f,%f)  rId=%d \n", receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);




	//Check incoming or outgoing ray
	rtBufferId<int, 3>& ib = internalRaysBuffer[receiverId];
	int prevRef = ib[receiverLaunchIndex];
	if (prevRef < 0) {
		//Incoming ray. Store number of reflections
		ib[receiverLaunchIndex] = reflections;
		++hitPayload.hits;
	}
	else {
		//Outgoing ray. Check reflections
		//Reinit the buffer
		ib[receiverLaunchIndex] = -1;
		if (prevRef == reflections) {
			//Ray has not been reflected within the receiver sphere, ignore it
			//rtPrintf("  rId=%d----> outgoing ray \n", receiverId);

			return;

		}
	}
	//float r = hit_attr.t; //Distance to origin
	//rtPrintf("HR. rx=(%f,%f,%f) radius=%f id=%u tx=%u\n", sphere.x,sphere.y,sphere.z, sphere.w, index.x, index.y);
	//rtPrintf("HR. HitPayload inte=%d \n", hitPayload.reflections);



	if (reflections == 0) {
		//This is a direct ray

		int old = atomicAdd(&receptionInfoBuffer[index].directHits, 1);
		//rtPrintf("before dh=%d,id=%u tx=%u receptionInfoBuffer[index].directHits=%d  i.x=%u i.y=%u, ray=(%f,%f,%f) t=%f \n", old, index.x, index.y, receptionInfoBuffer[index].directHits, receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, hit_attr.t);
		if (old > 0) {
			//rtPrintf("HR. already direct \n");
			//Already has a direct ray counted, just ignore
			return;
		}
		//Compute electric field. For direct rays, the distance is always between tx and rx
		float3 prx = make_float3(sphere.x, sphere.y, sphere.z);
		float3 ptx = tx_origin[launchIndex.z].origin;
		float d = length(prx - ptx);
		

		//rtPrintf("DR. Direct hit   i.x=%u i.y=%u ray=(%f,%f,%f) prx=(%f,%f,%f) ptx=(%f,%f,%f) rId=%d \n",  receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, prx.x, prx.y, prx.z, ptx.x, ptx.y, ptx.z, receiverId);
		//rtPrintf("HR. Ep.x=%f Ep.y=%f Eo=%f\n", receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, hitPayload.electricFieldAmplitude);
		float2 z = make_float2(0.0f, -k*d);
		//rtPrintf("HR. prx=(%f,%f,%f) k=%f d=%f prevTd=%f hitPayload.totalDistance=%f Eo=%f\n", prx.x, prx.y, prx.z, k, d, prevTd, hitPayload.totalDistance, hitPayload.electricFieldAmplitude);

		float2 zexp = complex_exp_only_imaginary(z);
		float2 E = sca_complex_prod((hitPayload.electricFieldAmplitude / d), zexp);

		float oldEx = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.x, E.x);
		float oldEy = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.y, E.y);
		//rtPrintf("DR. Direct hit   i.x=%u i.y=%u  Ep=(%f,%f) E=(%f,%f) En=(%f,%f) rId=%d \n", receiverLaunchIndex.x, receiverLaunchIndex.y, oldEx,oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);
		//Direct hit info log (to be used in external programs)
		
		rtPrintf("DH\t%u\t%u\t%u\t%f\t%f\t%f\t%f\t%f\t%f\t%d\n", receiverLaunchIndex.x, receiverLaunchIndex.y, receiverLaunchIndex.z, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);

		//receptionInfoBuffer[index].hasDirectContribution = true;
		//rtPrintf("%f\t%f\n", E.x, E.y);
	}
	else {

		//Reflected ray
		//Check for duplicates
		//index of buffer
		uint2 idf = make_uint2(hitPayload.faceId, receiverId);

		//Represent the number of reflections with a bit position in a 32 bit unsigned integer. Up to 31 reflections. A ray with higher number of reflections should have been killed by the closest hit at mesh
		uint refBit = 0u;
		//Set bit at position (reflections) to 1, so for instance reflections=2 means ... ... ... 00000100 (32 bit unsigned)
		refBit |= 1u << hitPayload.reflections;

		uint oldR = atomicOr(&facesBuffer[idf], refBit);
		//Check the value of the bit at position (reflections) 

		uint bit = (oldR >> hitPayload.reflections) & 1u;

		//Bit already set for the block, consider it a duplicate
		if (bit == 1) {
			//	rtPrintf(" duplicated ray reflections=%d i.x=%u i.y=%u with idb.x=%u idb.y=%u oldR=%u bit=%u rId=%d\n", hitPayload.reflections, receiverLaunchIndex.x, receiverLaunchIndex.y, idx.x, idx.y,oldR, bit, receiverId);
			return;
		}

		//rtPrintf("Reflected hit reflections=%d i.x=%u i.y=%u db=%d with idb.x=%u idb.y=%u oldR=%u bit=%u rId=%d \n", hitPayload.reflections, receiverLaunchIndex.x, receiverLaunchIndex.y, db[receiverLaunchIndex], idx.x, idx.y, oldR, bit, receiverId);
		atomicAdd(&receptionInfoBuffer[index].reflections, 1);

		float3 prx = make_float3(sphere.x, sphere.y, sphere.z);
		float3 ptx = ray_receiver.origin;
		float d = length(prx - ptx);
		//Compute electric field
		//rtPrintf("ref totalDistance=%f d=%f reflections=%d i.x=%u i.y=%u \n", hitPayload.totalDistance, d, hitPayload.reflections, receiverLaunchIndex.x, receiverLaunchIndex.y);
		d += prevTd; //totalDistance

		float2 z = make_float2(0.0f, -k*d);
		float2 zexp = complex_exp_only_imaginary(z);
		float2 Rzexp = complex_prod(hitPayload.prodReflectionCoefficient, zexp);


		float2 E = sca_complex_prod((hitPayload.electricFieldAmplitude / d), Rzexp);
		//rtPrintf("ref R=(%f,%f) z=(%f,%f) zepx(%f,%f) Rzexp=(%f,%f), E=(%f,%f) i.x=%u i.y=%u \n", hitPayload.prodReflectionCoefficient.x, hitPayload.prodReflectionCoefficient.y, z.x, z.y, zexp.x, zexp.y, Rzexp.x, Rzexp.y, E.x, E.y, receiverLaunchIndex.x, receiverLaunchIndex.y);

		float oldEx = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.x, E.x);
		float oldEy = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.y, E.y);
		//rtPrintf("HR. i.x=%u i.y=%u  Reflected hit  reflections=%d Ep=(%f,%f) E=(%f,%f) En=(%f,%f) rId=%d \n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);

		//rtPrintf("Old E=(%f.%f) New E=(%f,%f) i.x=%u i.y=%u \n", oldx, oldy, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverLaunchIndex.x, receiverLaunchIndex.y);
		//rtPrintf("%f\t%f\n", E.x, E.y);
		//Reflected hit info log (to be used in external programs)
		rtPrintf("F\t%u\t%u\t%u\t%f\t%f\t%f\t%f\t%f\t%f\t%d\t%f\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId, prevTd, length(prx - ptx));


	}

}
*/



RT_PROGRAM void closestHitReceiverFaceMin()
{

	//TODO: We do not check polarization between tx and rx. Can be done comparing payload polarization and receiver polarization

	//Do not end the ray, it can pass the reception sphere and reflect on a wall, inside or outside the receiver sphere
	//Store for use later
	float prevTd = hitPayload.totalDistance;

	//Update ray data
	hitPayload.totalDistance += hit_attr.t;
	hitPayload.hitPoint = ray_receiver.origin + hit_attr.t*ray_receiver.direction;
	hitPayload.nextDirection = ray_receiver.direction;

	
	//Check if ray is hitting his own tx (transmitter are also receiver usually) A transmitter cannot receive while it is transmitting, unless other channel is used.
	uint2 index = make_uint2(receiverId, receiverLaunchIndex.z);
	if (externalId == receiverLaunchIndex.z) {
		//Outgoing ray
		rtPrintf("External. id=%u tx=%u i.x=%u i.y=%u, ray=(%f,%f,%f) origin=(%f,%f,%f) t=%f rId=%d\n", index.x, index.y, receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, ray_receiver.origin.x, ray_receiver.origin.y, ray_receiver.origin.z, hit_attr.t, receiverId);

		return;
	}


	//uint2 index = make_uint2(receiverId, receiverLaunchIndex.z);//Index is given by transmitter and receiver ids
	int reflections = hitPayload.reflections;

	//Check incoming or outgoing ray
	rtBufferId<int, 3>& ib = internalRaysBuffer[receiverId];
	int prevRef = ib[receiverLaunchIndex];
	uint3 myrr = make_uint3(90, 90, 0);
	if (prevRef < 0) {
		//Incoming ray. Store number of reflections
		ib[receiverLaunchIndex] = reflections;
		++hitPayload.hits;
		
		//rtPrintf("IR\t%u\t%u\t%u\t%d\t%d\t%d\n", receiverLaunchIndex.x, receiverLaunchIndex.y, receiverLaunchIndex.z, prevRef, reflections, ib[receiverLaunchIndex]);
		
	}
	else {
		//Outgoing ray. Check reflections
		//Reinit the buffer
		ib[receiverLaunchIndex] = -1;
		

		
		if (prevRef == reflections) {
			//Ray has not been reflected within the receiver sphere, ignore it
			//rtPrintf("  rId=%d----> outgoing ray \n", receiverId);
			
			//rtPrintf("OR\t%u\t%u\t%u\t%d\t%d\n", receiverLaunchIndex.x, receiverLaunchIndex.y, receiverLaunchIndex.z, prevRef, reflections);
			
			return;

		}
	}
	//float r = hit_attr.t; //Distance to origin
	//rtPrintf("HR. rx=(%f,%f,%f) radius=%f id=%u tx=%u\n", sphere.x,sphere.y,sphere.z, sphere.w, index.x, index.y);
	//rtPrintf("HR. HitPayload inte=%d \n", hitPayload.reflections);



	if (reflections == 0) {
		
		//This is a direct ray

		int old = atomicAdd(&receptionInfoBuffer[index].directHits, 1);
		//rtPrintf("before dh=%d,id=%u tx=%u receptionInfoBuffer[index].directHits=%d  i.x=%u i.y=%u, ray=(%f,%f,%f) t=%f \n", old, index.x, index.y, receptionInfoBuffer[index].directHits, receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, hit_attr.t);
		if (old > 0) {
			//rtPrintf("HR. already direct \n");
			//Already has a direct ray counted, just ignore
			return;
		}
		//Compute electric field. For direct rays, the distance is always between tx and rx
		float3 prx = make_float3(sphere.x, sphere.y, sphere.z);
		float3 ptx = tx_origin[launchIndex.z].origin;
		float d = length(prx - ptx);
		//rtPrintf("DHd\t%u\t%u\t%u\t%f\t%f\t%f\t%f\t%f\t%d\t%d\n", receiverLaunchIndex.x, receiverLaunchIndex.y, receiverLaunchIndex.z, sphere.x, sphere.y, sphere.z, sphere.w,d, receiverId,externalId);
		//rtPrintf("DR. Direct hit   i.x=%u i.y=%u ray=(%f,%f,%f) prx=(%f,%f,%f) ptx=(%f,%f,%f) rId=%d \n",  receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, prx.x, prx.y, prx.z, ptx.x, ptx.y, ptx.z, receiverId);
		//rtPrintf("HR. Ep.x=%f Ep.y=%f Eo=%f\n", receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, hitPayload.electricFieldAmplitude);
		float2 z = make_float2(0.0f, -k*d);
		//rtPrintf("HR. prx=(%f,%f,%f) k=%f d=%f prevTd=%f hitPayload.totalDistance=%f Eo=%f\n", prx.x, prx.y, prx.z, k, d, prevTd, hitPayload.totalDistance, hitPayload.electricFieldAmplitude);

		float2 zexp = complex_exp_only_imaginary(z);
		float2 E = sca_complex_prod((hitPayload.electricFieldAmplitude / d), zexp);

		float oldEx = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.x, E.x);
		float oldEy = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.y, E.y);
		//rtPrintf("DHd\t%u\t%u\t%u\t%f\t%f\t%f\t%d\n", receiverLaunchIndex.x, receiverLaunchIndex.y, receiverLaunchIndex.z, oldEx, oldEy,d, receiverId);

		//rtPrintf("DR. Direct hit   i.x=%u i.y=%u  Ep=(%f,%f) E=(%f,%f) En=(%f,%f) rId=%d \n", receiverLaunchIndex.x, receiverLaunchIndex.y, oldEx,oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);
		//Direct hit info log (to be used in external programs)
		//rtPrintf("DH\t%u\t%u\t%u\t%f\t%f\t%f\t%f\t%f\t%f\t%d\t%d\n", receiverLaunchIndex.x, receiverLaunchIndex.y, receiverLaunchIndex.z, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId,externalId);

		//receptionInfoBuffer[index].hasDirectContribution = true;
		//rtPrintf("%f\t%f\n", E.x, E.y);
	}
	else {

		//Reflected ray
		//Check for duplicates: keep the ray closest to the center of the receiver
		
	
		uint3 idmd = make_uint3(reflections-1, hitPayload.faceId, receiverId);
		
		//Distance from ray line to receiver position
		//Line is defined by ray
		float3 prx = make_float3(sphere.x, sphere.y, sphere.z);
		float3 pd = prx - hitPayload.hitPoint;
		float u = dot(pd, ray_receiver.direction);
		float3 p3 = hitPayload.hitPoint + u*ray_receiver.direction;


		float dm = length(prx - p3)*1000000.0f;  //Multiply by 1000 000 to truncate later take 6 digits
		int dmt = __float2int_rz(dm);   //Truncate
		int oldd = atomicMin(&bufferMinD[idmd], dmt);
		
		if (oldd < dmt) {
			//our distance is greater,return
			//rtPrintf("FRF\t%u\t%u\t%u\t%f\t%d\t%d\t%f\t%f\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, dm, dmt, oldd, hitPayload.hitPoint.x, hitPayload.hitPoint.y, hitPayload.hitPoint.z);
			
			return;
		}
		//rtPrintf("T\t%u\t%u\t%u\t%u\t%f\t%d\t%d\t%f\t%f\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, hitPayload.faceId,dm, dmt, oldd, hitPayload.hitPoint.x, hitPayload.hitPoint.y, hitPayload.hitPoint.z);


		//rtPrintf("Reflected hit reflections=%d i.x=%u i.y=%u db=%d with idb.x=%u idb.y=%u oldR=%u bit=%u rId=%d \n", hitPayload.reflections, receiverLaunchIndex.x, receiverLaunchIndex.y, db[receiverLaunchIndex], idx.x, idx.y, oldR, bit, receiverId);
		atomicAdd(&receptionInfoBuffer[index].reflections, 1);

		
		float3 ptx = ray_receiver.origin;
		float d = length(prx - ptx);
		//Compute electric field
		//rtPrintf("ref totalDistance=%f d=%f reflections=%d i.x=%u i.y=%u \n", hitPayload.totalDistance, d, hitPayload.reflections, receiverLaunchIndex.x, receiverLaunchIndex.y);
		d += prevTd; //totalDistance

		float2 z = make_float2(0.0f, -k*d);
		float2 zexp = complex_exp_only_imaginary(z);
		float2 Rzexp = complex_prod(hitPayload.prodReflectionCoefficient, zexp);


		float2 E = sca_complex_prod((hitPayload.electricFieldAmplitude / d), Rzexp);
		/*
		if (receiverLaunchIndex.x==1118 && receiverLaunchIndex.y==900) {
			rtPrintf("ref R=(%f,%f) z=(%f,%f) zepx(%f,%f) Rzexp=(%f,%f), E=(%f,%f) i.x=%u i.y=%u \n", hitPayload.prodReflectionCoefficient.x, hitPayload.prodReflectionCoefficient.y, z.x, z.y, zexp.x, zexp.y, Rzexp.x, Rzexp.y, E.x, E.y, receiverLaunchIndex.x, receiverLaunchIndex.y);
		}
		*/
		
		
		//float2 Eprev = bufferMinE[idmd].E;
		//Update min buffer
		//bufferMinE[idmd].E = E;
		//Update min buffer
		float*  drx = &bufferMinE[idmd].E.x;
		float*  dry = &bufferMinE[idmd].E.y;
		float Eprevx = atomicExch(drx, E.x);
		float Eprevy = atomicExch(dry, E.y);
		float2 Eprev = make_float2(Eprevx, Eprevy);
		//rtPrintf("C\t%u\t%u\t%u\t%u\t%d\t%f\t%f\t%f\t%f\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, hitPayload.faceId,  dmt,  E.x, E.y, hitPayload.prodReflectionCoefficient.x, hitPayload.prodReflectionCoefficient.y, d);

		//rtPrintf("FF\t%u\t%u\t%u\t%u\t%f\t%d\t%d\t%f\t%f\t%f\t%f\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, hitPayload.faceId, dm, dmt, oldd, E.x, E.y, Eprev.x, Eprev.y, d);

		//Remove Electric field from previous minimum distance hit
		E -= Eprev; 

		//Update the receiver
		float oldEx = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.x, E.x);
		float oldEy = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.y, E.y);
		//rtPrintf("HR. i.x=%u i.y=%u  Reflected hit  reflections=%d Ep=(%f,%f) E=(%f,%f) En=(%f,%f) rId=%d \n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);

		//rtPrintf("Old E=(%f.%f) New E=(%f,%f) i.x=%u i.y=%u \n", oldx, oldy, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverLaunchIndex.x, receiverLaunchIndex.y);
		//rtPrintf("%f\t%f\n", E.x, E.y);
		//Reflected hit info log (to be used in external programs)
		//rtPrintf("F\t%u\t%u\t%u\t%u\t%f\t%f\t%f\t%f\t%f\t%f\t%d\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, hitPayload.faceId, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId, prevTd);


	}

}



//Mainly for debug purposes: The electric field values of all reflected hits are kept in a buffer and can be summed after launch.
RT_PROGRAM void closestHitReceiverFaceMinHoldReflections()
{

	//TODO: We do not check polarization between tx and rx. Can be done comparing payload polarization and receiver polarization

	//Do not end the ray, it can pass the reception sphere and reflect on a wall, inside or outside the receiver sphere
	//Store for use later
	float prevTd = hitPayload.totalDistance;

	//Update ray data
	hitPayload.totalDistance += hit_attr.t;
	hitPayload.hitPoint = ray_receiver.origin + hit_attr.t*ray_receiver.direction;
	hitPayload.nextDirection = ray_receiver.direction;

	//Check if ray is hitting his own tx (transmitter are also receiver usually) A transmitter cannot receive while it is transmitting, unless other channel is used.
	uint2 index = make_uint2(receiverId, receiverLaunchIndex.z);
	if (externalId == receiverLaunchIndex.z) {
		//Outgoing ray
		//rtPrintf("External. id=%u tx=%u i.x=%u i.y=%u, ray=(%f,%f,%f) origin=(%f,%f,%f) t=%f rId=%d\n", index.x, index.y, receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, ray_receiver.origin.x, ray_receiver.origin.y, ray_receiver.origin.z, hit_attr.t, receiverId);

		return;
	}


	//uint2 index = make_uint2(receiverId, receiverLaunchIndex.z);//Index is given by transmitter and receiver ids
	int reflections = hitPayload.reflections;

	//Check incoming or outgoing ray
	rtBufferId<int, 3>& ib = internalRaysBuffer[receiverId];
	int prevRef = ib[receiverLaunchIndex];
	if (prevRef < 0) {
		//Incoming ray. Store number of reflections
		ib[receiverLaunchIndex] = reflections;
		++hitPayload.hits;
	}
	else {
		//Outgoing ray. Check reflections
		//Reinit the buffer
		ib[receiverLaunchIndex] = -1;
		if (prevRef == reflections) {
			//Ray has not been reflected within the receiver sphere, ignore it
			//rtPrintf("  rId=%d----> outgoing ray \n", receiverId);

			return;

		}
	}
	//float r = hit_attr.t; //Distance to origin
	//rtPrintf("HR. rx=(%f,%f,%f) radius=%f id=%u tx=%u\n", sphere.x,sphere.y,sphere.z, sphere.w, index.x, index.y);
	//rtPrintf("HR. HitPayload inte=%d \n", hitPayload.reflections);



	if (reflections == 0) {
		//This is a direct ray

		int old = atomicAdd(&receptionInfoBuffer[index].directHits, 1);
		//rtPrintf("before dh=%d,id=%u tx=%u receptionInfoBuffer[index].directHits=%d  i.x=%u i.y=%u, ray=(%f,%f,%f) t=%f \n", old, index.x, index.y, receptionInfoBuffer[index].directHits, receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, hit_attr.t);
		if (old > 0) {
			//rtPrintf("HR. already direct \n");
			//Already has a direct ray counted, just ignore
			return;
		}
		//Compute electric field. For direct rays, the distance is always between tx and rx
		float3 prx = make_float3(sphere.x, sphere.y, sphere.z);
		float3 ptx = tx_origin[launchIndex.z].origin;
		float d = length(prx - ptx);
		//rtPrintf("DR. Direct hit   i.x=%u i.y=%u ray=(%f,%f,%f) prx=(%f,%f,%f) ptx=(%f,%f,%f) rId=%d \n",  receiverLaunchIndex.x, receiverLaunchIndex.y, ray_receiver.direction.x, ray_receiver.direction.y, ray_receiver.direction.z, prx.x, prx.y, prx.z, ptx.x, ptx.y, ptx.z, receiverId);
		//rtPrintf("HR. Ep.x=%f Ep.y=%f Eo=%f\n", receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, hitPayload.electricFieldAmplitude);
		float2 z = make_float2(0.0f, -k*d);
		//rtPrintf("HR. prx=(%f,%f,%f) k=%f d=%f prevTd=%f hitPayload.totalDistance=%f Eo=%f\n", prx.x, prx.y, prx.z, k, d, prevTd, hitPayload.totalDistance, hitPayload.electricFieldAmplitude);

		float2 zexp = complex_exp_only_imaginary(z);
		float2 E = sca_complex_prod((hitPayload.electricFieldAmplitude / d), zexp);

		float oldEx = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.x, E.x);
		float oldEy = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.y, E.y);
		//rtPrintf("DR. Direct hit   i.x=%u i.y=%u  Ep=(%f,%f) E=(%f,%f) En=(%f,%f) rId=%d \n", receiverLaunchIndex.x, receiverLaunchIndex.y, oldEx,oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);
		//Direct hit info log (to be used in external programs)
		rtPrintf("DH\t%u\t%u\t%u\t%f\t%f\t%f\t%f\t%f\t%f\t%d\n", receiverLaunchIndex.x, receiverLaunchIndex.y, receiverLaunchIndex.z, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);

		//receptionInfoBuffer[index].hasDirectContribution = true;
		//rtPrintf("%f\t%f\n", E.x, E.y);
	}
	else {

		//Reflected ray
		//Check for duplicates
		//Index of buffer is receiverLaunchIndex= receiverLaunchIndex.x, receiverLaunchIndex.y,txId
		uint3 idmd = make_uint3(reflections-1, hitPayload.faceId, receiverId);

		//Distance from ray line to receiver position
		//Line is defined by ray
		float3 prx = make_float3(sphere.x, sphere.y, sphere.z);
		float3 pd = prx - hitPayload.hitPoint;
		float u = dot(pd, ray_receiver.direction);
		float3 p3 = hitPayload.hitPoint + u*ray_receiver.direction;


		float dm = length(prx - p3)*1000000.0f;  //Multiply by 1000 000 to truncate later take 6 digits
		int dmt = __float2int_rz(dm);   //Truncate
		int oldd = atomicMin(&bufferMinD[idmd], dmt);
		if (oldd < dmt) {
			//our distance is greater,return
			//rtPrintf("FRF\t%u\t%u\t%u\t%f\t%d\t%d\t%f\t%f\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, dm, dmt, oldd, hitPayload.hitPoint.x, hitPayload.hitPoint.y, hitPayload.hitPoint.z);

			return;
		}
		//rtPrintf("FT\t%u\t%u\t%u\t%f\t%d\t%d\t%f\t%f\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, dm, dmt, oldd, hitPayload.hitPoint.x, hitPayload.hitPoint.y, hitPayload.hitPoint.z);


		//rtPrintf("Reflected hit reflections=%d i.x=%u i.y=%u db=%d with idb.x=%u idb.y=%u oldR=%u bit=%u rId=%d \n", hitPayload.reflections, receiverLaunchIndex.x, receiverLaunchIndex.y, db[receiverLaunchIndex], idx.x, idx.y, oldR, bit, receiverId);
		atomicAdd(&receptionInfoBuffer[index].reflections, 1);


		float3 ptx = ray_receiver.origin;
		float d = length(prx - ptx);
		//Compute electric field
		//rtPrintf("ref totalDistance=%f d=%f reflections=%d i.x=%u i.y=%u \n", hitPayload.totalDistance, d, hitPayload.reflections, receiverLaunchIndex.x, receiverLaunchIndex.y);
		d += prevTd; //totalDistance

		float2 z = make_float2(0.0f, -k*d);
		float2 zexp = complex_exp_only_imaginary(z);
		float2 Rzexp = complex_prod(hitPayload.prodReflectionCoefficient, zexp);


		float2 E = sca_complex_prod((hitPayload.electricFieldAmplitude / d), Rzexp);
		//rtPrintf("ref R=(%f,%f) z=(%f,%f) zepx(%f,%f) Rzexp=(%f,%f), E=(%f,%f) i.x=%u i.y=%u \n", hitPayload.prodReflectionCoefficient.x, hitPayload.prodReflectionCoefficient.y, z.x, z.y, zexp.x, zexp.y, Rzexp.x, Rzexp.y, E.x, E.y, receiverLaunchIndex.x, receiverLaunchIndex.y);



		//float2 Eprev = bufferMinE[idmd].E;
		//Update min buffer
		//bufferMinE[idmd].E = E;
		//Update min buffer
		float*  drx = &bufferMinE[idmd].E.x;
		float*  dry = &bufferMinE[idmd].E.y;
		float Eprevx = atomicExch(drx, E.x);
		float Eprevy = atomicExch(dry, E.y);
		float2 Eprev = make_float2(Eprevx, Eprevy);
		rtPrintf("FF\t%u\t%u\t%u\t%f\t%d\t%d\t%f\t%f\t%f\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, dm, dmt, oldd, E.x, E.y, Eprev.x, Eprev.y);

		//Remove Electric field from previous minimum distance hit
		/*E -= Eprev;

		//Update the receiver
		float oldEx = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.x, E.x);
		float oldEy = atomicAdd(&receptionInfoBuffer[index].sumRxElectricField.y, E.y);
		//rtPrintf("HR. i.x=%u i.y=%u  Reflected hit  reflections=%d Ep=(%f,%f) E=(%f,%f) En=(%f,%f) rId=%d \n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId);

		//rtPrintf("Old E=(%f.%f) New E=(%f,%f) i.x=%u i.y=%u \n", oldx, oldy, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverLaunchIndex.x, receiverLaunchIndex.y);
		//rtPrintf("%f\t%f\n", E.x, E.y);
		//Reflected hit info log (to be used in external programs)
		rtPrintf("F\t%u\t%u\t%u\t%f\t%f\t%f\t%f\t%f\t%f\t%d\t%f\t%f\n", receiverLaunchIndex.x, receiverLaunchIndex.y, reflections, oldEx, oldEy, E.x, E.y, receptionInfoBuffer[index].sumRxElectricField.x, receptionInfoBuffer[index].sumRxElectricField.y, receiverId, prevTd, length(prx - ptx));

		*/
	}

}



rtDeclareVariable(EMWavePayload, missPayload, rtPayload, );
RT_PROGRAM void miss()
{
	//rtPrintf("miss i.x=%u. iy=%u \n", receiverLaunchIndex.x, receiverLaunchIndex.y);
	missPayload.end = true;
}

